#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <float.h>

// 行优先存储
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
// 将一个指针转换为 float4 类型的指针并访问它指向的第一个元素
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

void cpuSgemm(
    float *A, float *B, float *C, const int M, const int N, const int K
){

    for (int i = 0;i < M;i++) {
        for (int j = 0;j < N;j++) {
            float sum = 0.0;
            for (int k = 0;k < K;k++) {
                sum += A[OFFSET(i, k, K)] * B[OFFSET(k, j, N)];
            }
            C[OFFSET(i, j, N)] = sum;
        }
    }

}

__global__ void naiveSgemm(
    float * __restrict__ A, float * __restrict__ B, float * __restrict__ C,
    const int M, const int N, const int K
){

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;

    if (m < M && n < N) {
        float sum = 0.0;
        #pragma unroll
        for (int k = 0; k < K; k++) {
            sum += A[OFFSET(m, k, K)] * B[OFFSET(k, n, N)];
        }
        C[OFFSET(m, n, N)] = sum;
    }

}

__global__ void mySgemmV1Aligned(
    float * __restrict__ A, float * __restrict__ B, float * __restrict__ C,
    const int M, const int N, const int K
) {
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[BM][BK];
    __shared__ float s_b[BK][BN];

    float r_c[TM][TN] = {0.0};              // 用于暂存C矩阵的数据，存放在寄存器中

    int load_a_smem_m = tid >> 1;           //  tid / 2
    int load_a_smem_k = (tid & 1) << 2;     //  (tid % 2) * 4
    int load_b_smem_k = tid >> 5;           //  tid / 32
    int load_b_smem_n = (tid & 31) << 2;    //  (tid % 32) * 4

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {
        // block内的线程分工把数据加载到shared memory中，即s_a、s_b
        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        // FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(A[load_a_gmem_addr]);

        // 只在边界情况下进行检查，避免所有加载都进行边界判断
        if (load_a_gmem_k + 3 < K && load_a_gmem_m < M) {
            FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(A[load_a_gmem_addr]);
        } else {
            // printf("Thread %d: load_a_gmem_k = %d, load_a_gmem_m = %d, load_a_gmem_addr = %d\n", tid, load_a_gmem_k, load_a_gmem_m, load_a_gmem_addr);
            float4 a_val;
            a_val.x = (load_a_gmem_m < M && load_a_gmem_k < K) ? A[load_a_gmem_addr] : 0.0f;
            a_val.y = (load_a_gmem_m < M && load_a_gmem_k + 1 < K) ? A[load_a_gmem_addr + 1] : 0.0f;
            a_val.z = (load_a_gmem_m < M && load_a_gmem_k + 2 < K) ? A[load_a_gmem_addr + 2] : 0.0f;
            a_val.w = (load_a_gmem_m < M && load_a_gmem_k + 3 < K) ? A[load_a_gmem_addr + 3] : 0.0f;
            FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = a_val;
        }

        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        // FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(B[load_b_gmem_addr]);

        

        if (load_b_gmem_n + 3 < N && load_b_gmem_k < K) {
            FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(B[load_b_gmem_addr]);
        } else {
            float4 b_val;
            b_val.x = (load_b_gmem_n < N && load_b_gmem_k < K) ? B[load_b_gmem_addr] : 0.0f;
            b_val.y = (load_b_gmem_n + 1 < N && load_b_gmem_k < K) ? B[load_b_gmem_addr + 1] : 0.0f;
            b_val.z = (load_b_gmem_n + 2 < N && load_b_gmem_k < K) ? B[load_b_gmem_addr + 2] : 0.0f;
            b_val.w = (load_b_gmem_n + 3 < N && load_b_gmem_k < K) ? B[load_b_gmem_addr + 3] : 0.0f;
            FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = b_val;
        }

        __syncthreads(); //等待所有线程的数据全部加载到shared memory中


        // 每个线程计算TM * TN 子矩阵的结果，并存储到r_c数组寄存器里
        #pragma unroll
        for (int k = 0; k < BK; k++) {
            #pragma unroll
            for (int m = 0; m < TM; m++) {
                #pragma unroll
                for (int n = 0; n < TN; n++) {
                    r_c[m][n] += s_a[ty * TM + m][k] * s_b[k][tx * TN + n];     //注意坐标的计算
                }
            }
        }

        __syncthreads(); 
    }

    // 把计算好的TM * TN子矩阵数组r_c存储到对应C矩阵中
    // int store_c_gemm_m = by * BM + ty * TM;
    #pragma unroll
    for (int i = 0;i < TM; i++) {
        // store_c_gemm_m++;
        int store_c_gmem_m = by * BM + ty * TM + i;
        #pragma unroll
        for (int j = 0; j < TN; j += 4) {
            int store_c_gmem_n = bx * BN + tx * TN + j;
            int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
            // FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][j]);

            if (store_c_gmem_n + 3 < N && store_c_gmem_m < M) {
                FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][j]);
            }
            else {
                if (store_c_gmem_m < M) {
                    for (int k = 0; k < 4; k++) {
                        if (store_c_gmem_n + k < N) {
                            C[store_c_gmem_addr + k] = r_c[i][j + k];
                        }
                    }
                }

            }

        }
    }



}


__global__ void mySgemmV2Aligned(
    float * __restrict__ A, float * __restrict__ B, float * __restrict__ C,
    const int M, const int N, const int K
) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[BK][BM];   // s_a逆转过来，以便取数据计算时可以连续取值
    __shared__ float s_b[BK][BN];

    float r_load_a[4];
    float r_load_b[4];
    float r_comp_a[TM];     //  存储从s_a取出的TM长度的向量
    float r_comp_b[TN];     //  存储从s_b取出的TN长度的向量

    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {

        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        if (load_a_gmem_m < M && load_a_gmem_k + 3 < K){
            FLOAT4(r_load_a[0]) = FLOAT4(A[load_a_gmem_addr]);
        }
        else {
            r_load_a[0] = (load_a_gmem_m < M && load_a_gmem_k < K) ? A[load_a_gmem_addr] : 0.0f;
            r_load_a[1] = (load_a_gmem_m < M && load_a_gmem_k + 1 < K) ? A[load_a_gmem_addr + 1] : 0.0f;
            r_load_a[2] = (load_a_gmem_m < M && load_a_gmem_k + 2 < K) ? A[load_a_gmem_addr + 2] : 0.0f;
            r_load_a[3] = (load_a_gmem_m < M && load_a_gmem_k + 3 < K) ? A[load_a_gmem_addr + 3] : 0.0f;
        }

        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        if (load_b_gmem_n + 3 < N && load_b_gmem_k < K) {
            FLOAT4(r_load_b[0]) = FLOAT4(B[load_b_gmem_addr]);
        }
        else {
            r_load_b[0] = (load_b_gmem_n < N && load_b_gmem_k < K) ? B[load_b_gmem_addr] : 0.0f;
            r_load_b[1] = (load_b_gmem_n + 1 < N && load_b_gmem_k < K) ? B[load_b_gmem_addr + 1] : 0.0f;
            r_load_b[2] = (load_b_gmem_n + 2 < N && load_b_gmem_k < K) ? B[load_b_gmem_addr + 2] : 0.0f;
            r_load_b[3] = (load_b_gmem_n + 3 < N && load_b_gmem_k < K) ? B[load_b_gmem_addr + 3] : 0.0f;
        }


        s_a[load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);

        __syncthreads();

        #pragma unroll
        for (int tk = 0; tk < BK; tk++) {

            // 从共享内存取出两个向量
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[tk][ty * TM / 2]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[tk][tx * TN / 2]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[tk][tx * TN / 2 + BN / 2]);

            // 计算外积，注意这里的矩阵内存位置不是连续的，不能直接写回C
            #pragma unroll
            for (int tm = 0; tm < TM; tm++) {
                #pragma unroll
                for (int tn = 0; tn < TN; tn++) {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }

        }
        
        __syncthreads();
    }

    // 把r_c矩阵根据空间变换写回矩阵C
    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);

        if (store_c_gmem_m < M) {
            if (store_c_gmem_n + 3 < N) {
                FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);

                if (store_c_gmem_n + 3 + BN  / 2 < N) {
                    FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
                }
                else if (store_c_gmem_n + BN / 2 < N) {
                    for (int k = 0; k < 4; k++) {
                        if (store_c_gmem_n + k + BN / 2 < N) {
                            C[store_c_gmem_addr + BN / 2 + k] = r_c[i + TM / 2][k];
                       }
                    }
                }
            }
            else {
                for (int k = 0; k < 4; k++) {
                    if (store_c_gmem_n + k < N) {
                        C[store_c_gmem_addr + k] = r_c[i][k];
                    }
                }

            }
        }
        // FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
        // FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
    }

    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + BM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);

        if (store_c_gmem_m < M) {
            if (store_c_gmem_n + 3 < N) {
                FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);

                if (store_c_gmem_n + 3 + BN  / 2 < N) {
                    FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
                }
                else if (store_c_gmem_n + BN / 2 < N) {

                    for (int k = 0; k < 4; k++) {
                        if (store_c_gmem_n + k + BN / 2 < N) {
                            C[store_c_gmem_addr + BN / 2 + k] = r_c[i + TM / 2][k];
                       }
                    }

                }
            }
            else {

                for (int k = 0; k < 4; k++) {
                    if (store_c_gmem_n + k < N) {
                        C[store_c_gmem_addr + k] = r_c[i][k];
                    }
                }

            }
        }
    

        // FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
        // FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
    }
}

__global__ void mySgemmV3Aligned(
    float * __restrict__ A, float * __restrict__ B, float * __restrict__ C,
    const int M, const int N, const int K
) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    // 使用双倍的share memory来预取数据，
    // 在计算数据之前加载下一次循环用到的数据（从 global Memory 加载到 Shared Memory)
    // GPU无法乱序执行，必须在计算之前就进行数据的加载
    __shared__ float s_a[2][BK][BM];   
    __shared__ float s_b[2][BK][BN];

    float r_load_a[4];
    float r_load_b[4];
    float r_comp_a[TM];     //  存储从s_a取出的TM长度的向量
    float r_comp_b[TN];     //  存储从s_b取出的TN长度的向量

    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    // 第一次把数据写进share memory中
    {
        int load_a_gmem_k = load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(r_load_a[0]) = FLOAT4(A[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(B[load_b_gmem_addr]);

        s_a[0][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[0][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[0][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[0][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[0][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
    }

    __syncthreads();
    
    // 循环次数减少一次，bk可以看做是加载第几次循环需要的数据到share memory中。
    // 循环内其实计算的是第 bk - 1 次的加载的数据
    for (int bk = 1; bk < (K + BK - 1) / BK ; bk++) {

        int smem_sel = (bk - 1) & 1;   // 当前循环计算需要使用的share memory序号
        int smem_next = bk & 1;

        // 把下一次循环用到的数据从global memory中加载到寄存器中（会不会寄存器不够用啊）
        // 这里使用的LDG指令进行load数据，不会影响后续的运算指令的发射执行
        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(r_load_a[0]) = FLOAT4(A[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(B[load_b_gmem_addr]);

        // 还有这里的同步指令不能使用了，我们希望加载与计算能够并行执行

        // 计算预取的数据
        #pragma unroll
        for (int tk = 0; tk < BK; tk++) {
            // 从共享内存取出两个向量
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

            // 计算外积，注意这里的矩阵内存位置不是连续的，不能直接写回C
            #pragma unroll
            for (int tm = 0; tm < TM; tm++) {
                #pragma unroll
                for (int tn = 0; tn < TN; tn++) {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }
        }

        // 把加载的数据从寄存器中写入共享内存中
        // 这部分的STS指令会等待LDG指令写回后再继续发射执行，所以不能放在计算部分之前
        s_a[smem_next][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[smem_next][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[smem_next][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[smem_next][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[smem_next][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
        
        __syncthreads();
    }

    // 计算最后一次循环
    int smem_sel = ((K + BK - 1) / BK - 1) & 1;  
    #pragma unroll
    for (int tk = 0; tk < BK; tk++) {
        // 从共享内存取出两个向量
        FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2]);
        FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
        FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2]);
        FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

        // 计算外积，注意这里的矩阵内存位置不是连续的，不能直接写回C
        #pragma unroll
        for (int tm = 0; tm < TM; tm++) {
            #pragma unroll
            for (int tn = 0; tn < TN; tn++) {
                r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
            }
        }
    }


    // 把r_c矩阵根据空间变换写回矩阵C
    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
        FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
    }
    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + BM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
        FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
    }
}

void printMatrix(float *M, int m, int n) {
    for(int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%12.6f  ", M[OFFSET(m, n, m)]);
        }
        printf("\n");
    }
}

float testCublasMaxError(const int M, const int N, const int K) {

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    
    h_a = (float *)malloc(M * K * sizeof(float));
    h_b = (float *)malloc(K * N * sizeof(float));
    h_c = (float *)malloc(M * N * sizeof(float));
    h_d_c = (float *)malloc(M * N * sizeof(float));
    hipMalloc(&d_a, M * K * sizeof(float));
    if (d_a == NULL) {
        // 处理错误，例如打印错误信息并退出
        printf("CUDA Malloc failed for d_a\n");
        exit(EXIT_FAILURE);
    }
    hipMalloc(&d_b, N * K * sizeof(float));
    if (d_b == NULL) {
        // 处理错误，例如打印错误信息并退出
        printf("CUDA Malloc failed for d_b\n");
        exit(EXIT_FAILURE);
    }
    hipMalloc(&d_c, M * N * sizeof(float));
    if (d_c == NULL) {
        // 处理错误，例如打印错误信息并退出
        printf("CUDA Malloc failed for d_c\n");
        exit(EXIT_FAILURE);
    }
    
    srand(time(0));

    for (int i = 0;i < M * K;i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0;i < N * K;i++)
        h_b[i] = rand() / float(RAND_MAX);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * K * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    float cublas_alpha = 1.0;
    float cublas_beta = 0.0;

    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &cublas_alpha, d_b, N, d_a, K, &cublas_beta, d_c, N);

    hipMemcpy(h_d_c, d_c,  M * N * sizeof(float), hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0;i < M * N;i++) {
        float error = abs(h_d_c[i] - h_c[i]);
        // 可能会出现极小的值，导致数据溢出，生成NaN
        if (max_error != max_error || error != error) {
            max_error = -NAN;
        }
        else
            max_error = max(max_error, error);
    }

    //释放空间
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_d_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(cublas_handle);

    return max_error;
}

float testCublasPerformance(const int M, const int N, const int K, const int repeat) {

    int size_a = M * K * sizeof(float);
    int size_b = N * K * sizeof(float);
    int size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;

    hipError_t cudaStat;

    cudaStat = hipMalloc(&d_a, size_a);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_a with error: %s\n", hipGetErrorString(cudaStat));
        return -1;
    }
    cudaStat = hipMalloc(&d_b, size_b);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_b with error: %s\n", hipGetErrorString(cudaStat));
        return -1;
    }
    cudaStat = hipMalloc(&d_c, size_c);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_c with error: %s\n", hipGetErrorString(cudaStat));
        return -1;
    }

    srand(time(0));

    float *h_a = (float *)malloc(M * K * sizeof(float));
    float *h_b = (float *)malloc(K * N * sizeof(float));

    for (int i = 0;i < M * K;i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0;i < N * K;i++)
        h_b[i] = rand() / float(RAND_MAX);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    float cublas_alpha = 1.0;
    float cublas_beta =  0.0;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    for (int i = 0; i < repeat; i++) {
        // hipblasSgemm默认认为矩阵是列优先的
        // C^t = B^t A^t , 这里的C^t是列优先的，因此这是C的行优先
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &cublas_alpha, d_b, N, d_a, K, &cublas_beta, d_c, N);
    }

    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    hipblasDestroy(cublas_handle);

    return sec;


}

float testMaxError(
    void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K
) {
    size_t size_a = M * K * sizeof(float);      // 27 * 66536 *4Bytes
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    // hipMalloc(&d_a, size_a);
    hipError_t err = hipMalloc(&d_a, size_a);
    if (err != hipSuccess) {
        printf("hipMalloc failed for d_a: %s\n", hipGetErrorString(err));
        // handle the error
    }

    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = rand() / float(RAND_MAX);
    hipMemset(d_c, 15, size_c);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    hipError_t cuda_err = hipGetLastError();
    if (cuda_err != hipSuccess) {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(cuda_err));
    }
    
    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;


}

float testPerformance(
    void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat
) {
    int size_a = M * K * sizeof(float);
    int size_b = N * K * sizeof(float);
    int size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;

    
    hipError_t cudaStat = hipGetLastError();  // 清除之前的错误状态

    cudaStat = hipMalloc(&d_a, size_a);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_a with error: %s\n", hipGetErrorString(cudaStat));
        return -1;
    }
    cudaStat = hipMalloc(&d_b, size_b);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_b with error: %s\n", hipGetErrorString(cudaStat));
        return -1;
    }
    cudaStat = hipMalloc(&d_c, size_c);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_c with error: %s\n", hipGetErrorString(cudaStat));
        return -1;
    }

    srand(time(0));
    float *h_a = (float *)malloc(M * K * sizeof(float));
    float *h_b = (float *)malloc(K * N * sizeof(float));
    for (int i = 0;i < M * K;i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0;i < N * K;i++)
        h_b[i] = rand() / float(RAND_MAX);
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    // float cublas_alpha = 1.0;
    // float cublas_beta =  0.0;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    for (int i = 0; i < repeat; i++) {
        // hipblasSgemm默认认为矩阵是列优先的
        // C^t = B^t A^t , 这里的C^t是列优先的，因此这是C的行优先
        gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    }

    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    // hipblasDestroy(cublas_handle);

    return sec;
}


int main() {
    const int M_list[16] = {27, 128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[16] = {65536, 128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[16] = {1152, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};
    const int outer_repeat = 5, inner_repeat = 1;
    const int TEST_NUM = 3;

    // {
    //     printf("\nKernerl = cublas \n");

    //     {
    //         const int M = 512, N = 512, K = 512;

    //         float max_error = testCublasMaxError(M, N, K);
    //         printf("Max Error = %f\n", max_error);
    //     }

    //     {


    //         for (int i = 0;i < TEST_NUM;i++) {
    //             const int M = M_list[i];
    //             const int N = N_list[i];
    //             const int K = K_list[i];

    //             double max_sec = 0.0;
    //             double min_sec = DBL_MAX;
    //             double total_sec = 0.0;

    //             for (int j = 0; j < outer_repeat; j++) {
    //                 double this_sec = testCublasPerformance(M, N, K, inner_repeat);
    //                 max_sec = max(max_sec, this_sec);
    //                 min_sec = min(min_sec, this_sec);
    //                 total_sec += this_sec;
    //             }

    //             double avg_sec = total_sec / outer_repeat;
    //             double avg_Gflops = ((double) M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

    //             printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    //         }
    //     }
    // }

    // {
    //     printf("\nKernel = naiveSgemm\n");

    //     const int BM = 32, BN = 32;
    //     void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) = 
    //         naiveSgemm;
        
    //     {
    //         const int M = 512, N = 512, K = 512;
    //         dim3 blockDim(BN, BM);
    //         dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
    //         float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
    //         printf("Max Error = %f\n", max_error);
    //     }

    //     {


    //         for (int i = 0;i < TEST_NUM;i++) {
    //             const int M = M_list[i];
    //             const int N = N_list[i];
    //             const int K = K_list[i];

    //             dim3 blockDim(BN, BM);
    //             dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

    //             double max_sec = 0.0;
    //             double min_sec = DBL_MAX;
    //             double total_sec = 0.0;

    //             for (int j = 0; j < outer_repeat; j++) {
    //                 double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
    //                 max_sec = max(max_sec, this_sec);
    //                 min_sec = min(min_sec, this_sec);
    //                 total_sec += this_sec;
    //             }

    //             double avg_sec = total_sec / outer_repeat;
    //             double avg_Gflops = ((double) M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

    //             printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    //         }
    //     }
    // }

    {
        printf("\nKernel = mySgemmV1Aligned\n");

        const int BM = 128, BN = 128, TM = 8, TN = 8;
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) = 
            mySgemmV1Aligned;
        
        {
            const int M = 512, N = 512, K = 512;
            dim3 blockDim(BN / TN, BM / TM);
            dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
            float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
            printf("Max Error = %f\n", max_error);


        }

        {
            

            for (int i = 0;i < TEST_NUM;i++) {
                const int M = M_list[i];
                const int N = N_list[i];
                const int K = K_list[i];

                dim3 blockDim(BN / TN, BM / TM);
                dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double) M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }

    {
        printf("\nKernel = mySgemmV2Aligned\n");

        const int BM = 128, BN = 128, TM = 8, TN = 8;
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) = 
            mySgemmV2Aligned;
        
        {
            const int M = 512, N = 512, K = 512;
            dim3 blockDim(BN / TN, BM / TM);
            dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
            float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        {

            for (int i = 0;i < TEST_NUM;i++) {
                const int M = M_list[i];
                const int N = N_list[i];
                const int K = K_list[i];

                dim3 blockDim(BN / TN, BM / TM);
                dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double) M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }

    {
        printf("\nKernel = mySgemmV3Aligned\n");

        const int BM = 128, BN = 128, TM = 8, TN = 8;
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) = 
            mySgemmV3Aligned;
        
        {
            const int M = 512, N = 512, K = 512;
            dim3 blockDim(BN / TN, BM / TM);
            dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
            float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        {

            for (int i = 0;i < TEST_NUM;i++) {
                const int M = M_list[i];
                const int N = N_list[i];
                const int K = K_list[i];

                dim3 blockDim(BN / TN, BM / TM);
                dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double) M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }

}