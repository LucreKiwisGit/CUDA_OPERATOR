#include <assert.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <mma.h>
#include <stdio.h>

using namespace nvcuda;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define MAX(a, b) ((a) >= (b) ? (a) : (b))
#define MIN(a, b) ((a) >= (b) ? (b) : (a))
#define ABS(x) ((x) >= 0 ? (x) : -(x))

const int WARP_SIZE = 32;
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

bool is_close(float input, float other, float rtol, float atol) { return ABS(input - other) <= atol + rtol * ABS(other); }

void cpu_conv(float *in, float *out, float *cpu_kernel, int IC, int IH, int IW, int OC, int OH, int OW, int KH, int KW) {
    int out_pos, in_pos, kernel_pos;
    for (int oc = 0; oc < OC; ++oc) {
        for (int i = 0; i < OH; ++i) {
            for (int j = 0; j < OW; ++j) {
                float val = 0;
                out_pos = oc * OH * OW + OFFSET(i, j, OW);
                for (int ic = 0; ic < IC; ++ic) {
                    for (int ii = 0; ii < KH; ++ii) {
                        for (int jj = 0; jj < KW; ++jj) {
                            if (i + ii >= IH || j + jj >= IW) continue;
                            in_pos = ic * IH * IW + OFFSET(i + ii, j + jj, IW);
                            kernel_pos = oc * IC * KH * KW + ic * KH * KW + OFFSET(ii, jj, KW);
                            val += in[in_pos] * cpu_kernel[kernel_pos];
                        }
                    }
                }
                out[out_pos] = val;
            }
        }
    }
}

template <const int KH, const int KW, const int WARPS_PER_BLOCK>
__global__ void implicit_gemm_conv(__half *input, __half *output, __half *kernel, const int N, const int IC, const int IH, const int IW, const int OC, const int OH, const int OW) {
    const int UNROLLED_KERNEL_SIZE = KH * KW;          // 每个卷积核单通道展开后的大小
    const int SLICE_SIZE = IC * UNROLLED_KERNEL_SIZE;  // 每个卷积核展开后的大小

    const int WMMA_INPUT_TILE_SIZE = WMMA_M * WMMA_K;
    const int WMMA_FILTER_TILE_SIZE = WMMA_K * WMMA_N;

    int GEMM_M = N * OH * OW;
    int GEMM_N = OC;
    int GEMM_K = SLICE_SIZE;

    const int out_n_stride = OH * OW;
    const int out_h_stride = OW;
    const int out_w_stride = 1;

    const int in_n_stride = IC * IH * IW;
    const int in_c_stride = IH * IW;
    const int in_h_stride = IW;
    const int in_w_stride = 1;

    const int kernel_n_stride = IC * KH * KW;
    const int kernel_c_stride = KH * KW;
    const int kernel_h_stride = KW;
    const int kernel_w_stride = 1;

    // 一个 block 中有 256 个线程，分成 256/32=8 个 warp
    const int global_warp_id_x = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;    // 每个 warp 在 x 方向的全局 id
    const int global_warp_id_y = (blockIdx.y * blockDim.y + threadIdx.y);                // 每个 warp 在 y 方向的全局 id
    const int block_warp_id_x = threadIdx.x / WARP_SIZE;                                 // 每个 warp 在所属 block 中 x 方向的 id
    const int block_warp_id_y = threadIdx.y;                                             // 每个 warp 在所属 block 中 y 方向的 id
    const int intra_warp_thread_id = threadIdx.x % WARP_SIZE;                            // 每个 thread 在 warp 中的 id
    const int num_warps_x = blockDim.x / WARP_SIZE;                                      // 每个 block 在 x 方向上的 warp 个数
    const int block_warp_id_linear = (block_warp_id_y * num_warps_x) + block_warp_id_x;  // 每个 warp 在所属 block 中的线性 id (0 to WARPS_PER_BLOCK - 1)

    // 共享内存中存放的是其对应 block 中 WARPS_PER_BLOCK 个 warp 计算需要的数据
    __shared__ __half smem_input_tile[WARPS_PER_BLOCK * WMMA_M * WMMA_K];
    __shared__ __half smem_weight_tile[WARPS_PER_BLOCK * WMMA_K * WMMA_N];

    // 声明 tensor core 计算需要的数据容器
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, __half> acc_frag;
    wmma::fill_fragment(acc_frag, __float2half(0.0f));

    // 每个 warp 需要的计算数据在共享内存中的位置
    __half *input_tile_start = smem_input_tile + (block_warp_id_linear * WMMA_INPUT_TILE_SIZE);
    __half *weight_tile_start = smem_weight_tile + (block_warp_id_linear * WMMA_FILTER_TILE_SIZE);

    #pragma unroll
    for (int i = 0; i < GEMM_K; i += WMMA_K) {
        int a_row = global_warp_id_x * WMMA_M;  // 每个 warp 对应的输入 tile 在输入矩阵中的行起始索引
        int a_col = i;                          // 每个 warp 对应的输入 tile 在输入矩阵中的列起始索引
        int b_row = i;                          // 每个 warp 对应的权重 tile 在权重矩阵中的行起始索引
        int b_col = global_warp_id_y * WMMA_N;  // 每个 warp 对应的权重 tile 在权重矩阵中的列起始索引

        // 读取输入矩阵
        #pragma unroll
        for (int j = intra_warp_thread_id; j < WMMA_INPUT_TILE_SIZE; j += WARP_SIZE) {  // 每个线程在 16x16 的输入 tile 中的 id
            int rel_slice_row = j / WMMA_K;                                             // 每个线程此时需要搬运的数据在 16x16 的输入 tile 中的行索引
            int abs_slice_row = a_row + rel_slice_row;                                  // 每个线程此时需要搬运的数据在输入矩阵中的全局行索引
            int abs_slice_col = a_col + (j % WMMA_K);                                   // 每个线程此时需要搬运的数据在输入矩阵中的全局列索引

            int n = abs_slice_row / out_n_stride;                                    // 每个线程此时需要搬运的数据在输出 feature map 中的 batch 索引
            int p = (abs_slice_row % out_n_stride) / out_h_stride;                   // 每个线程此时需要搬运的数据在输出 feature map 中的行索引
            int q = ((abs_slice_row % out_n_stride) % out_h_stride) / out_w_stride;  // 每个线程此时需要搬运的数据在输出 feature map 中的列索引

            int offsets[3] = {0, 1, 2};

            int y = p + offsets[(abs_slice_col % UNROLLED_KERNEL_SIZE) / KW];  // 每个线程此时需要搬运的数据在输入 feature map 中的列索引
            int x = q + offsets[(abs_slice_col % UNROLLED_KERNEL_SIZE) % KW];  // 每个线程此时需要搬运的数据在输入 feature map 中的行索引
            int c = abs_slice_col / UNROLLED_KERNEL_SIZE;                      // 每个线程此时需要搬运的数据在输入 feature map 中的通道索引

            if (x >= 0 && x < IW && y >= 0 && y < IH) {  // 防止边缘处越界
                int idx = n * in_n_stride + c * in_c_stride + y * in_h_stride + x * in_w_stride;
                input_tile_start[j] = input[idx];
            } else {
                input_tile_start[j] = __float2half(0.f);
            }
        }

        // 读取权重
        #pragma unroll
        for (int j = intra_warp_thread_id; j < WMMA_FILTER_TILE_SIZE; j += WARP_SIZE) {  // 每个线程在 16x16 的输入 tile 中的 id
            int rel_slice_row = j / WMMA_K;                                              // 每个线程此时需要搬运的数据在 16x16 的输入 tile 中的行索引
            int abs_slice_row = b_row + rel_slice_row;                                   // 每个线程此时需要搬运的数据在权重矩阵中的全局行索引
            int abs_slice_col = b_col + (j % WMMA_K);                                    // 每个线程此时需要搬运的数据在权重矩阵中的全局列索引

            int k = abs_slice_col;                                // 每个线程此时需要搬运的数据在卷积核中的卷积核索引
            int c = abs_slice_row / UNROLLED_KERNEL_SIZE;         // 每个线程此时需要搬运的数据在卷积核中的通道索引
            int r = (abs_slice_row % UNROLLED_KERNEL_SIZE) / KW;  // 每个线程此时需要搬运的数据在卷积核中的列索引
            int s = (abs_slice_row % UNROLLED_KERNEL_SIZE) % KW;  // 每个线程此时需要搬运的数据在卷积核中的行索引

            int idx = k * kernel_n_stride + c * kernel_c_stride + r * kernel_h_stride + s * kernel_w_stride;
            weight_tile_start[j] = kernel[idx];
        }

        // 使用 tensor core 完成矩阵乘法
        if (a_row < GEMM_M && a_col < GEMM_K && b_row < GEMM_K && b_col < GEMM_N) {
            wmma::load_matrix_sync(a_frag, input_tile_start, WMMA_K);
            wmma::load_matrix_sync(b_frag, weight_tile_start, WMMA_N);
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    int c_col = global_warp_id_y * WMMA_N;  // 每个 warp 对应的输入 tile 在输出矩阵中的行起始索引
    int c_row = global_warp_id_x * WMMA_M;  // 每个 warp 对应的输入 tile 在输出矩阵中的列起始索引

    // 写回输出矩阵
    if (c_row < GEMM_M && c_col < GEMM_N) {
        wmma::store_matrix_sync(output + (c_row + c_col * GEMM_M), acc_frag, GEMM_M, wmma::mem_col_major);
    }
}

int main() {
    const int PADDING_H = 1;
    const int PADDING_W = 1;
    const int STRIDE_H = 1;
    const int STRIDE_W = 1;
    const int KH = 3;
    const int KW = 3;
    const int WARPS_PER_BLOCK = 8;

    const int N = 1;
    const int IC = 8;
    const int IH = 64;
    const int IW = 64;
    const int OC = 16;
    const int OH = (IH - KH + 2 * PADDING_H) / STRIDE_H + 1;
    const int OW = (IW - KW + 2 * PADDING_W) / STRIDE_W + 1;

    int GEMM_M = N * OH * OW;
    int GEMM_N = OC;
    assert(GEMM_M % WMMA_M == 0);
    assert(GEMM_N % WMMA_N == 0);
    assert(N == 1);

    int input_size = N * IC * IH * IW;
    int output_size = OC * OH * OW;
    int kernel_size = OC * IC * KH * KW;
    float *cpu_input, *cpu_output, *cpu_kernel, *cuda_output;
    cpu_input = (float *)malloc(input_size * sizeof(float));
    cpu_output = (float *)malloc(output_size * sizeof(float));
    cpu_kernel = (float *)malloc(kernel_size * sizeof(float));
    cuda_output = (float *)malloc(output_size * sizeof(float));

    for (int i = 0; i < input_size; ++i) {
        cpu_input[i] = 1.0 * i / input_size;
    }
    for (int i = 0; i < kernel_size; ++i) {
        cpu_kernel[i] = 1.0 * i / kernel_size;
    }
    for (int i = 0; i < output_size; ++i) {
        cpu_output[i] = 0.0;
        cuda_output[i] = 0.0;
    }

    /* ---- CPU BEGIN ---- */
    cpu_conv(cpu_input, cpu_output, cpu_kernel, IC, IH, IW, OC, OH, OW, KH, KW);
    /* ---- CPU END ---- */

    /* ---- GPU BEGIN ---- */
    hipError_t err;

    __half *gpu_input_fp16, *gpu_kernel_fp16, *gpu_output_fp16;
    hipMalloc(&gpu_input_fp16, input_size * sizeof(__half));
    hipMalloc(&gpu_output_fp16, output_size * sizeof(__half));
    hipMalloc(&gpu_kernel_fp16, kernel_size * sizeof(__half));

    // convert data to fp16
    __half *cpu_input_fp16, *cpu_output_fp16, *cpu_kernel_fp16, *cuda_output_fp16;
    cpu_input_fp16 = (__half *)malloc(input_size * sizeof(__half));
    cpu_output_fp16 = (__half *)malloc(output_size * sizeof(__half));
    cpu_kernel_fp16 = (__half *)malloc(kernel_size * sizeof(__half));
    cuda_output_fp16 = (__half *)malloc(output_size * sizeof(__half));

    for (int i = 0; i < input_size; ++i) cpu_input_fp16[i] = __float2half(cpu_input[i]);
    for (int i = 0; i < kernel_size; ++i) cpu_kernel_fp16[i] = __float2half(cpu_kernel[i]);
    for (int i = 0; i < output_size; ++i) cpu_output_fp16[i] = __float2half(0.f);

    hipMemcpy(gpu_input_fp16, cpu_input_fp16, input_size * sizeof(__half), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("hipMemcpy input failed: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(gpu_kernel_fp16, cpu_kernel_fp16, kernel_size * sizeof(__half), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("hipMemcpy kernel failed: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(gpu_output_fp16, cpu_output_fp16, output_size * sizeof(__half), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("hipMemcpy output failed: %s\n", hipGetErrorString(err));
    }

    dim3 dim_block(128, 2);
    dim3 dim_grid((GEMM_M + (WMMA_M * dim_block.x / 32 - 1)) / (WMMA_M * dim_block.x / 32), (GEMM_N + WMMA_N * dim_block.y - 1) / (WMMA_N * dim_block.y));
    assert(dim_block.y * dim_block.x / WARP_SIZE == WARPS_PER_BLOCK);

    implicit_gemm_conv<KH, KW, WARPS_PER_BLOCK><<<dim_grid, dim_block>>>(gpu_input_fp16, gpu_output_fp16, gpu_kernel_fp16, N, IC, IH, IW, OC, OH, OW);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("execute kernel function: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(cuda_output_fp16, gpu_output_fp16, output_size * sizeof(__half), hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("hipMemcpy output failed: %s\n", hipGetErrorString(err));
    }

    for (int i = 0; i < output_size; ++i) {
        float cuda_value = __half2float(cuda_output_fp16[i]);
        float cpu_value = cpu_output[i];
        if (!is_close(cuda_value, cpu_value, 1e-2, 1e-3)) {
            printf("WRONG VALUE: %.5f | %.5f at %d\n", cpu_output[i], cuda_value, i);
            break;
        }
    }
    /* ---- GPU END ---- */

    printf("done!\n");
}