#include "hip/hip_runtime.h"
#include "operators.h"
#include <hip/hip_runtime_api.h>
#include "utils.h"
#include <stdio.h>
#include "convs2d.h"
#include <memory>
#include <omp.h>
#include "verify.h"
#include <chrono>

// 自定义删除器， 用于释放CUDA内存
struct CudaDeleter {
    template <typename T>
    void operator()(T *ptr) const {
        hipFree(ptr);
    }
};

void implicit_gemm_fp16_test() {

    int caseSize = 6;

    int N[caseSize] = {1, 64, 256, 16, 32, 2, 2};
    int C[caseSize] = {3, 256, 192, 256, 256, 1280, 960};
    int H[caseSize] = {12, 14,   14,  26,   14,  16,  64};
    int W[caseSize] = {12, 14,   14,  26,   14,  16,  64};
    int K[caseSize] = {1, 256,  192,   512,  256, 1280,   32};
    int R[caseSize] = {3, 3,    3,    3,    3,   3,   3};
    int S[caseSize] = {3, 3,    3,    3,    3,   3,   3};
    int P[caseSize] = {1, 1,    1,    1,    1,   1,   1};
    int Q[caseSize] = {1, 1,    1,    1,    1,   1,   1};
    int U[caseSize] = {1, 1,    1,    1,    1,   1,   1};
    int V[caseSize] = {1, 1,    1,    1,    1,   1,   1};

    for (int i = 0; i < caseSize; i++) {
        int n = N[i];
        int c = C[i];
        int h = H[i];
        int w = W[i];
        int k = K[i];
        int r = R[i];
        int s = S[i];
        int p = P[i];
        int q = Q[i];
        int u = U[i];
        int v = V[i];

        int OH = (w - r + 2 * p) / u + 1;
        int OW = (w - s + 2 * q) / v + 1;
        // int M = k;
        // int N = n * OH * OW;
        // int K = c * r * s;

        double flopsPerConv = 2.0 * n * k * c * r * s * OH * OW;

        // 分配空间
        auto input = std::make_unique<float[]>(n * c * h * w);
        auto weight = std::make_unique<float[]>(k * c * r * s);
        auto bias = std::make_unique<float[]>(k);
        auto output = std::make_unique<float[]>(n * k * OH * OW);
        auto output_benchmark = std::make_unique<float[]>(n * k * OH * OW);
        // float *input = (float *)malloc(n * c * h * w * sizeof(float));
        // float *weight = (float *)malloc(k * c * r * s * sizeof(float));
        // float *bias = (float *)malloc(k * sizeof(float));
        // float *output = (float *)malloc(n * k * OH * OW * sizeof(float));
        // float *output_benchmark = (float *)malloc(n * k * OH * OW * sizeof(float));

        std::unique_ptr<float, CudaDeleter> input_device(nullptr, CudaDeleter());
        std::unique_ptr<float, CudaDeleter> weight_device(nullptr, CudaDeleter());
        std::unique_ptr<float, CudaDeleter> bias_device(nullptr, CudaDeleter());
        std::unique_ptr<float, CudaDeleter> output_device(nullptr, CudaDeleter());
        
        hipError_t err;
        err = hipMalloc((void **)&input_device, n * c * h * w * sizeof(float));
        if (err != hipSuccess) {
            printf("hipMalloc failed: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        err = hipMalloc((void **)&weight_device, k * c * r * s * sizeof(float));
        if (err != hipSuccess) {
            printf("hipMalloc failed: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        err = hipMalloc((void **)&bias_device, k * sizeof(float));
        if (err != hipSuccess) {
            printf("hipMalloc failed: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        err = hipMalloc((void **)&output_device, n * k * OH * OW * sizeof(float));
        if (err != hipSuccess) {
            printf("hipMalloc failed: %s\n", hipGetErrorString(err));
            exit(-1);
        }

        // float *input_device, *weight_device, *bias_device, *output_device;
        // hipMalloc((void **)&input_device, n * c * h * w * sizeof(float));
        // hipMalloc((void **)&weight_device, k * c * r * s * sizeof(float));
        // hipMalloc((void **)&bias_device, k * sizeof(float));
        // hipMalloc((void **)&output_device, n * k * OH * OW * sizeof(float));

        // random input
        for (int ii = 0; ii < n * c * h * w; ii++) {
            input[ii] = (float)rand() / RAND_MAX;
        }

        for (int ii = 0; ii < k * c * r * s; ii++) {
            weight[ii] = (float)rand() / RAND_MAX;
        }

        for (int ii = 0; ii < k; ii++) {
            bias[ii] = (float)rand() / RAND_MAX;
        }

        // // 固定输入，用于debug
        // for (int ii = 0; ii < n * c * h * w; ii++) {
        //     input[ii] = (float)ii / 100;
        // }

        // for (int ii = 0; ii < k * c * r * s; ii++) {
        //     weight[ii] = (float)ii / 100;
        // }

        // for (int ii = 0; ii < k; ii++) {
        //     bias[ii] = (float)ii / 100;
        // }

        for (int ii = 0; ii < n * k * OH * OW; ii++) {
            output[ii] = (float)rand() / RAND_MAX;
            output_benchmark[ii] = (float)rand() / RAND_MAX;
        }

        hipMemcpy(input_device.get(), input.get(), n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(weight_device.get(), weight.get(), k * c * r * s * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(bias_device.get(), bias.get(), k * sizeof(float), hipMemcpyHostToDevice);

        // parameter 
        param_t param;

        param.input = input_device.get();
        param.weight = weight_device.get();
        param.bias = bias_device.get();
        param.output = output_device.get();
        param.input_host = input.get();
        param.weight_host = weight.get();
        param.bias_host = bias.get();
        param.output_host = output.get();
        param.output_benchmark = output_benchmark.get();
        param.n = n;
        param.c = c;
        param.h = h;
        param.w = w;
        param.k = k;
        param.kh = r;
        param.kw = s;
        param.pad_h = p;
        param.pad_w = q;
        param.stride_h = u;
        param.stride_w = v;
        param.Oh = OH;
        param.Ow = OW;

        printf("================Beigin=========================\n");
        printf("%2d %2d %2d %2d %d %d %2d\n", n, h, w, c, r, s, k);

        
        // warm up
        launch_implgemm(param);
        // direct_conv2dCuDNN(param);

        hipMemcpy(output.get(), output_device.get(), n * k * OH * OW * sizeof(float), hipMemcpyDeviceToHost);

        // 验证正确率
        
        auto start_ref = std::chrono::steady_clock::now();
        omp_set_num_threads(8);
        // direct_conv2dCpu(param);
        direct_conv2dCuDNN(param);
        hipMemcpy(output_benchmark.get(), output_device.get(), n * k * OH * OW * sizeof(float), hipMemcpyDeviceToHost);

        auto end_ref = std::chrono::steady_clock::now();


        int error = 0;
        for (int ii = 0; ii < n * k * OH * OW; ii++) {
            if (abs(output[ii] - output_benchmark[ii]) > abs(output[ii]) * 0.00001) {
                printf("error, postion:%d, cudnnvalue:%.8f, implGEMMvalue:%.8f\n", ii, output_benchmark[ii], output[ii]);
                error++;
                if (error > 0) {
                    break;
                }
            }
        }

        launch_implgemm(param);

        
        auto time_elapsed_ref = std::chrono::duration_cast<std::chrono::milliseconds>(end_ref - start_ref);
        double gflops = flopsPerConv / (time_elapsed_ref.count() / 1000.0) / 1e9 ;

        // printf(" time: %ld ms\n", time_elapsed_ref.count());
        printf("Cudnn Time per convolution: %ld ms\n", time_elapsed_ref.count()    );
        printf("Cudnn Performance :%f GFlops\n",  gflops);
        printf("================finish,error:%d=========================\n", error);
    
        

        // 测量 CUDA 内核函数的执行时间
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, nullptr);  // 显式使用nullptr代替0

        float time_elapsed = 0.0;

        int iternum = 10;
        for (int i = 0; i < iternum; i++) {
            launch_implgemm(param);
        }

        hipEventRecord(stop, nullptr);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_elapsed, start, stop);  // time_elapsed 以毫秒为单位

        hipEventDestroy(start);
        hipEventDestroy(stop);

        // 计算每次卷积的平均时间和GFlops
        float timePerConv = time_elapsed / iternum;
        gflops = flopsPerConv / (timePerConv / 1000.0) / 1e9;

        printf("n: %2d, h: %2d, w: %2d, c: %2d, r: %d, s: %d, k: %2d\n", n, h, w, c, r, s, k);
        printf("MyImplGEMM Time per convolution: %f ms\n", timePerConv);
        printf("MyImplGEMM Performance: %f GFlops\n", gflops);

    }
}


